#include "hip/hip_runtime.h"
#include "SMC2samplerGraph.hpp"


__global__  // CHILD KERNEL FIRST LEVEL
void MetropolisHastingsReject(  hiprandState_t *state,
                                const cuData &data,
                                const Graph &graph,
                                double *theta,
                                double *theta_new,
                                double *x_theta,
                                double *x_theta_new,            // [(T_current + 1)] new sampled time series up to t = T_current
                                double *mlh,
                                double *mlh_new,
                                double *x_particles,
                                double *x_particles_new,
                                double *w_x_particles,
                                double *w_x_particles_new)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
    {
        #if VERBOSE
            printf("[MHR] Inside MH reject\n");
        #endif
        hiprandState local_state = state[tid];
        size_t delta_t = abs(graph.current - graph.first);
        // Accept or reject new time series using MH
        double u = hiprand_uniform_double(&local_state);
        if (*mlh_new / *mlh >= u)
        {
            theta[0] = theta_new[0];
            theta[1] = theta_new[1];
            *mlh = *mlh_new;
            if (graph.direction == -1)
            {
                Map<VectorXd> x_theta_new_vec(x_theta_new, (delta_t + 1));
                VectorXd x_theta_flipped = x_theta_new_vec.reverse();
                memcpy(x_theta, x_theta_flipped.data(), sizeof(double) * (delta_t + 1));
            }
            else
            {
                memcpy(x_theta, x_theta_new, sizeof(double) * (delta_t + 1));
            }
            memcpy(x_particles, x_particles_new, sizeof(double) * data.data.N_x);
            memcpy(w_x_particles, w_x_particles_new, sizeof(double) * data.data.N_x);
            // printf("Accepted.\n");
        }
        state[tid] = local_state;
        hipFree(theta_new);
        hipFree(x_theta_new);
        hipFree(x_particles_new);
        hipFree(w_x_particles_new);
        hipFree(mlh_new);
    }
}
/**
 * Particle rejuvination
*/
__global__  // PARENT KERNEL
void MarginalMetropolisHastings(hiprandState_t *global_state_theta,
                                hiprandState_t *global_state_x,
                                const Graph &graph,
                                double *theta,                  // [2 x N_theta]
                                double *x,                      // [N x N_theta]
                                double *mlh,                    // [N_theta x 1]
                                double *x_particles,            // [N_x x N_theta]
                                double *w_x_particles,          // [N_x x N_theta]
                                const cuData &data)
{
    // x, x_particles and w_x_particles have size N but full of zeros. I need to consider only the 
    // first T_current rows/columns. In fact the PF PMMH regenerates samples only up to t = T_current.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data.data.N_theta)
    {
        #if VERBOSE
        // if (tid == 0)
        {
            printf("[MMH] tid: %d\n", tid);
        }
        #endif
        hiprandState local_state = global_state_theta[tid];
        Map<MatrixXd> theta_mat(theta, 2, data.data.N_theta);
        Map<MatrixXd> x_mat(x, data.data.N, data.data.N_theta);
        Map<MatrixXd> x_particles_mat(x_particles, data.data.N_x, data.data.N_theta);
        Map<MatrixXd> w_x_particles_mat(w_x_particles, data.data.N_x, data.data.N_theta);
        // hipStream_t streamMMH;
        // hipStreamCreateWithFlags(&streamMMH, hipStreamNonBlocking);
    
        // Sample new candidate theta for the new time series
        double *theta_new = new double[2];
        // skipahead(50, &local_state);
        theta_new[0] = 1.0 + hiprand_normal_double(&local_state);
        theta_new[1] = 1.0 + hiprand_normal_double(&local_state);
        global_state_theta[tid] = local_state;
        
        double *mlh_new;
        double *x_theta_new, *x_particles_new, *w_x_particles_new;
        
        size_t delta_t = abs(graph.current - graph.first);
        hipMalloc((void**)&x_theta_new, sizeof(double) * (delta_t + 1));
        hipMalloc((void**)&x_particles_new, sizeof(double) * data.data.N_x);
        hipMalloc((void**)&w_x_particles_new, sizeof(double) * data.data.N_x);
        hipMalloc((void**)&mlh_new, sizeof(double));
        // ParticleFilterPMMH<<<1, 1, 0, cudaStreamTailLaunch>>>(theta_new, T_current, data, global_state_x, mlh_new, x_theta_new, x_particles_new, w_x_particles_new);
        ParticleFilterPMMH<<<1, 1, sizeof(double) * data.data.N * data.data.N, cudaStreamTailLaunch>>>
                        (theta_new, graph, data, global_state_x, mlh_new, x_theta_new, x_particles_new, w_x_particles_new);
        MetropolisHastingsReject<<<1, 1, 0, cudaStreamTailLaunch>>>
                        (global_state_theta, data, graph, theta_mat.col(tid).data(), theta_new, x_mat.col(tid).data(), x_theta_new, &mlh[tid], mlh_new, 
                        x_particles_mat.col(tid).data(), x_particles_new, w_x_particles_mat.col(tid).data(), w_x_particles_new);
        // __syncthreads();
        // hipStreamDestroy(streamMMH);

    }
}

__global__  // CHILD KERNEL SECOND LEVEL
void PropagateState(hiprandState_t *global_state,
                    const int T_current,            // Time at which prediction is made
                    double *x_t,                    // [N_x x 1] x-particles at T_current
                    double *w_x_t,                  // [N_x x 1] x-weights
                    double *L,                      // [N x N] sqrt(K) st dev matrix
                    const cuData &data)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data.data.N_x)
    {
        #if VERBOSE
        if (1)//(tid == 0)
        {
            printf("[PROP] tid %d. Time %d.\n", tid, T_current);
        }
        #endif
        // printf("N is %d\n", data.data.N);
        // printf("T-th row of L is:\n");
        // printf("L: %f\t%f\n", L[0], L[99]);
        // print_matrix(data.data.N, data.data.N, L, data.data.N);
        // printf("x particles at time %d\n", T_current);
        // print_matrix(data.data.N_x, 1, x_t, data.data.N_x);
        hiprandState local_state = global_state[tid];
        Map<VectorXd> x_t_vec(x_t, data.data.N_x);
        Map<VectorXd> w_x_t_vec(w_x_t, data.data.N_x);
        Map<MatrixXd> L_mat(L, data.data.N, data.data.N);
    
        VectorXd rand_var(data.data.N);
        // skipahead(100, global_state);
        for (int ii = 0; ii < data.data.N; ii++)
        {
            rand_var(ii) = hiprand_normal_double(&local_state);
        }
        // x_t_vec(tid) = L_mat.col(T_current).transpose() * rand_var;
        x_t_vec(tid) = L_mat.row(T_current) * rand_var;
        w_x_t_vec(tid) = exp( -0.5 * log(2 * M_PI * data.data.Rnoise) - 0.5 * pow(data.data.Y[T_current] - x_t_vec(tid), 2) / data.data.Rnoise );
        // printf("[PROP] tid %d. Time: %d. w_x_t is %e; x_t is %f; y is %f\n", tid, T_current, w_x_t[tid], x_t[tid], data.data.Y[T_current]);
        global_state[tid] = local_state;
    }
}

/**
 * Metropolis Resempling algorithm for creating ancestors
*/
__global__
void MetropolisResampling(  hiprandState_t *global_state, 
                            double *weights,                // [N_particles x 1]
                            const int N_particles, 
                            const int iters, 
                            int* ancestors)                 // [N_particles x 1]
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Forse solo thread.x? Se no non parallelizza sui blocchi
    // printf("blockDim = %d; blockIdx = %d; threadIdx = %d\n", blockDim.x, blockIdx.x, threadIdx.x);
    if (tid < N_particles)
    {
        Map<VectorXi> ancestors_vec(ancestors, N_particles);
        ancestors_vec.setZero();
        #if VERBOSE
        if (tid == 0)
        {
            printf("[MR] tid is: %d. N_particles is %d. Iters is %d\n", tid, N_particles, iters);
        }
        #endif
        hiprandState local_state = global_state[tid];

        int k = tid;
        for (int t = 0; t < iters; t++)
        {
            double u = hiprand_uniform_double(&local_state);
            double jd = hiprand_uniform_double(&local_state);
            jd *= (N_particles - 1 + 0.999999);
            int j = (int)trunc(jd);
            if ( u <= weights[j] / weights[k] && !isnan(weights[j] / weights[k]) )
            {
                k = j;
            }
            

        }
        ancestors_vec(tid) = k;
        global_state[tid] = local_state;
    }
}

__global__
void PermutateStatesAndWeights(const cuData &data, double *x_t, double *w_x_t, const int* a)
{
    extern __shared__ double tmp_buffer[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data.data.N_x)
    {
        #if VERBOSE
        if (tid == 0)
        {
            printf("[PERM_STATES] tid = %d\n", tid);
        }
        #endif
        
        Map<VectorXd> w_x_t_vec(w_x_t, data.data.N_x);
        if (w_x_t_vec.sum() < 1e-200)
        {
            // If weights are all zero, re-initialize them as 1 / N_x and return.
            w_x_t[tid] = 1.0 / data.data.N_x;
            // printf("[PERM_STATES] a contains: %d %d %d %d.\tw_x[tid]: %e; sum(w_x): %e\n", a[0], a[1], a[2], a[3], w_x_t[tid], w_x_t_vec.sum());
            return;
        }
        // printf("[PERM_STATES] a contains: %d %d %d %d.\tw_x[tid]: %e; sum(w_x): %e\n", a[0], a[1], a[2], a[3], w_x_t[tid], w_x_t_vec.sum());

        memcpy(&tmp_buffer, x_t, sizeof(double) * data.data.N_x);
        x_t[tid] = tmp_buffer[a[tid]];
        memcpy(&tmp_buffer, w_x_t, sizeof(double) * data.data.N_x);
        w_x_t[tid] = tmp_buffer[a[tid]];
        // hipMemcpyAsync(x_t_old, x_t, sizeof(double) * data.data.N_x, hipMemcpyDeviceToDevice);
        // hipMemcpyAsync(w_x_t_old, w_x_t, sizeof(double) * data.data.N_x, hipMemcpyDeviceToDevice);
        __syncthreads();
        
        // printf("[PERM_STATES] w_x_perm[%d]: %e; w_x_t_old[%d]: %e\n", tid, w_x_t[tid], tid, w_x_t_old[tid]);
    }
}

__global__
void FinalizePFPMMH(const cuData &data,
                    const Graph &graph,
                    double *x,              // [N_x x (T_current + 1)] All x-particles for each time instant
                    double *w_x,            // Matrix of N_x weights for (T_current + 1) steps
                    double *mlh_hat,        // [1 x 1] Marginal LH referred to this theta vector
                    double *x_hat,          // [(T_current + 1) x 1] Time series for this theta vector
                    double *x_particles,    // [N_x x 1]
                    double *w_x_particles)  // Vector of only the last N_x weights
{
    // Transform the N_x x-particles from t=1:T_current into x_hat. 
    // Compute mlh_hat and save last generated x-particles and weights.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
    {
        #if VERBOSE
        printf("[FIN_PMMH] tid: %d\n", tid);
        #endif
        // printf("[FIN_PMMH] w_x is:\n");
        // print_matrix(data.data.N_x, (T_current + 1), w_x, data.data.N_x);
        // Average the states over the weights and return time series, 
        size_t delta_t = abs(graph.current - graph.first);
        Map<MatrixXd> x_mat(x, data.data.N_x, (delta_t + 1));
        Map<MatrixXd> w_x_mat(w_x, data.data.N_x, (delta_t + 1));
        // Map<const::VectorXd> mlh_hat_vec(mlh_hat, data.data.N_theta);
        // printf("[FIN_PMMH] x is\n");
        // print_matrix(data.data.N_x, (T_current + 1), x, data.data.N_x);
        VectorXd w_x_summed = w_x_mat.colwise().sum(); // Along columns: [1 x N_x] vector
        // *mlh_hat = (w_x_mat.colwise().sum() / data.data.N_x).prod();
        *mlh_hat = (w_x_summed / data.data.N_x).prod();
        // Map<VectorXd> x_hat_vec(x_hat, data.data.N);
        Map<VectorXd> x_hat_vec(x_hat, delta_t + 1);
        Map<VectorXd> x_particles_vec(x_particles, data.data.N_x);
        Map<VectorXd> w_x_particles_vec(w_x_particles, data.data.N_x);
        
        VectorXd w_x_tmp = w_x_summed.array().inverse();
        MatrixXd w_hat_normalized = w_x_mat * w_x_tmp.asDiagonal();
        // printf("[FIN_PMMH] w_x_normalized is:\n");
        // print_matrix(data.data.N_x, (T_current + 1), w_hat_normalized.data(), data.data.N_x);
        x_hat_vec = ( w_hat_normalized.array() * x_mat.array() ).colwise().sum();
        // x_hat_vec = ( w_x.array() / w_x.colwise().sum().array() * x.array() ).colwise().sum();
        x_particles_vec = x_mat.rightCols(1);
        w_x_particles_vec = w_x_mat.rightCols(1);
        // x_particles_vec = x_mat.col(T_current);
        // w_x_particles_vec = w_x_mat.col(T_current);
        // printf("x_particles:\n");
        // print_matrix(data.data.N_x, 1, x_particles, data.data.N_x);
        hipFree(x);
        hipFree(w_x);
    }
}

__global__
void cudaFreePF(hipStream_t stream, double *L, int *a)
{
    // Just hipFree() the matrices used in PF kernel
    hipStreamDestroy(stream);
    hipFree(L);
    hipFree(a);
}

/**
 * Particle Filter Metropolis Hastings (PMMH)
*/
__global__  // CHILD KERNEL FIRST LEVEL
void ParticleFilterPMMH(double *theta,                  // [2 x 1] One theta vector out of N_theta theta vectors
                        const Graph &graph, 
                        const cuData &data, 
                        hiprandState_t *global_state, 
                        double *mlh_hat,                // [1 x 1] Marginal LH referred to this particular time series
                        double *x_hat,                  // [(T_current + 1) x 1] Time series referred to this theta vector
                        double *x_particles,            // [N_x x 1] Last x-particles used for propagation
                        double *w_x_particles)          // [N_x x 1]
{
    // This kernel runs for each N_theta theta.
    extern __shared__ double K[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
    {
        #if VERBOSE
        printf("[PMMH] tid: %d; %d --> %d; T_curr: %d\n", tid, graph.first, graph.last, graph.current);
        #endif
        // (Re)Initialize
        double *x, *w_x;
        size_t delta_t = abs(graph.current - graph.first);
        hipMalloc((void**)&x, sizeof(double) * data.data.N_x * (delta_t + 1));
        hipMalloc((void**)&w_x, sizeof(double) * data.data.N_x * (delta_t + 1));
        
        Map<MatrixXd> x_mat(x, data.data.N_x, (delta_t + 1));     // x_hat up to t = T_current
        Map<MatrixXd> w_x_mat(w_x, data.data.N_x, (delta_t + 1));
        x_mat.setZero();
        w_x_mat.setZero();
        w_x_mat.col(0) = VectorXd::Ones(data.data.N_x) / data.data.N_x;

        // K can be built incrementally at each time instant: K = computeKernel(system_x(0:T_current), system_x(0:T_current), theta[0], theta[1]);

        // double *K;
        // hipMalloc((void**)&K, sizeof(double) * data.data.N * data.data.N);
        computeKernel(data.data.X, data.data.N, data.data.X, data.data.N, theta[0], theta[1], K);
        // Map<MatrixXd> K_mat(K, data.data.N, data.data.N);
        // K_mat.setIdentity();
        // printf("K is:\n");
        // print_matrix(data.data.N, data.data.N, K, data.data.N);


        double *L_tmp;
        hipMalloc((void**)&L_tmp, sizeof(double) * data.data.N * data.data.N);
        // hipMalloc((void**)&L, sizeof(double) * data.data.N * data.data.N);
        cuCholesky(K, data.data.N, L_tmp);
        // hipFree(K);
        // Map<MatrixXd> L_tmp_mat(L_tmp, data.data.N, data.data.N);
        // MatrixXd L = L_tmp_mat.transpose();
        Map<MatrixXd> L(L_tmp, data.data.N, data.data.N);

        // printf("L is:\n");
        // print_matrix(data.data.N, data.data.N, L.data(), data.data.N);
        VectorXd rand_var(data.data.N_x);
        hiprandState local_state = global_state[tid];
        double f_0 = L(0,0) * hiprand_normal_double(&local_state);
        for (int ii = 0; ii < data.data.N_x; ii++)
        {
            rand_var(ii) = hiprand_normal_double(&local_state) + f_0;
        }
        x_mat.col(0) = rand_var;
        global_state[tid] = local_state;
        int *a;
        hipMalloc((void**)&a, sizeof(int) * data.data.N_x);
        // VectorXi a(data.data.N_x);
        hipStream_t streamPMMH;
        hipStreamCreateWithFlags(&streamPMMH, hipStreamNonBlocking);
        // __syncthreads();

        for (int t = 1; t < delta_t + 1; t++)
        {
            // printf("[PMMH] Time instant %d of %d. Launching propagate kernel from thread %d...\n", t, T_current, tid);
            
            // PropagateState<<<1, 16, 0, cudaStreamFireAndForget>>>(global_state, t, x_mat.col(t).data(), w_x_mat.col(t).data(), L.data(), data); // THIS WORKS!
            PropagateState<<<1, data.data.N_x, 0, streamPMMH>>>(global_state, t, x_mat.col(t).data(), w_x_mat.col(t).data(), L.data(), data);
            
            // Metropolis resampling (__device__ kernel)

            MetropolisResampling<<<1, data.data.N_x, 0, streamPMMH>>>(global_state, w_x_mat.col(t).data(), data.data.N_x, data.data.B, a);
            
            // PermutateStatesAndWeights<<<1, data.data.N_x, 0, streamPMMH>>>(data, x_mat.col(t).data(), w_x_mat.col(t).data(), a);
            PermutateStatesAndWeights<<<1, data.data.N_x, sizeof(double) * data.data.N_x, streamPMMH>>>(data, x_mat.col(t).data(), w_x_mat.col(t).data(), a); // With shared memory
        }

        // FinalizePFPMMH<<<1, 1, 0, cudaStreamTailLaunch>>>(data, x_mat.data(), w_x_mat.data(), mlh_hat, x_hat, x_particles, w_x_particles);
        FinalizePFPMMH<<<1, 1, 0, streamPMMH>>>(data, graph, x_mat.data(), w_x_mat.data(), mlh_hat, x_hat, x_particles, w_x_particles);
        cudaFreePF<<<1, 1, 0, cudaStreamTailLaunch>>>(streamPMMH, L.data(), a);
        // hipStreamDestroy(streamPMMH);
        // __syncthreads();
    }

}

__global__
void FinalizePF(const cuData &data,
                const int T_next,
                double *x_predicted,            // PF one-step prediction
                double *w_x_predicted,          // PF predicted last-step weights
                double *mlh_hat,                // PF estimated mlh until step t updated to t+1
                double *x_hat_theta,            // Updated trajectory at t+1 for each theta
                double *x_particles,            // N_x particles for next iteration
                double *w_x_particles)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
    {
        #if VERBOSE
        printf("[FIN_PF] tid: %d\n", tid);
        #endif
        // Function referred to a single theta pair.
        Map<const::VectorXd> w_x_pred_vec(w_x_predicted, data.data.N_x);
        Map<const::VectorXd> x_pred_vec(x_predicted, data.data.N_x);
        Map<VectorXd> x_hat_theta_mat(x_hat_theta, data.data.N);
        
        *mlh_hat = *mlh_hat / data.data.N_x * w_x_pred_vec.sum();
        double x_t_plus_one = ((w_x_pred_vec.array() * x_pred_vec.array()) / w_x_pred_vec.sum()).sum();
        x_hat_theta_mat(T_next) = x_t_plus_one;
        
        memcpy(x_particles, x_predicted, sizeof(double) * data.data.N_x);
        memcpy(w_x_particles, w_x_predicted, sizeof(double) * data.data.N_x);
        hipFree(x_predicted);
        hipFree(w_x_predicted);
    }
}

/**
 * Bootstrap Particle Filter for one-step ahead prediction
*/

__global__
void ParticleFilter(double *theta,                  // [2 x N_theta] Matrix with all thetas
                    const int T_next, 
                    const cuData &data, 
                    hiprandState_t *global_state, 
                    double *mlh_hat,                // [N_theta x 1] Marginal LH for each theta 
                    double *x_hat,                  // [N x N_theta] Time series for each theta
                    double *x_particles,            // [N_x x N_theta] N_x particles obtained in the last step
                    double *w_x_particles)
{
    // This prediction step has to be done for each theta. States and weights are N x N_theta matrices for this reason!
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < data.data.N_theta)
    {
        #if VERBOSE
        if (tid == 0)
        {
            printf("[PF] tid: %d\n", tid);
        }
        #endif
        // x_particles are the N_x last-step particles for each N_theta
        Map<MatrixXd> x_particles_mat(x_particles, data.data.N_x, data.data.N_theta);
        Map<MatrixXd> w_x_particles_mat(w_x_particles, data.data.N_x, data.data.N_theta);
        Map<MatrixXd> theta_mat(theta, 2, data.data.N_theta);
        // Map<VectorXd> system_x(data.data.X, data.data.N);

        // K can be built incrementally at each time instant: K = computeKernel(system_x(0:T_current), system_x(0:T_current), theta[0], theta[1]);
        // MatrixXd K = computeKernel(system_x, system_x, theta[0], theta[1]) + 1e-6 * MatrixXd::Identity(data.data.N, data.data.N);
        
        double *K;
        hipMalloc((void**)&K, sizeof(double) * data.data.N * data.data.N);
        computeKernel(data.data.X, data.data.N, data.data.X, data.data.N, theta_mat(0, tid), theta_mat(1, tid), K);

        double *L_tmp;
        hipMalloc((void**)&L_tmp, sizeof(double) * data.data.N * data.data.N);
        cuCholesky(K, data.data.N, L_tmp);
        hipFree(K);
        // MatrixXd L(data.data.N, data.data.N);
        Map<MatrixXd> L(L_tmp, data.data.N, data.data.N);
        // MatrixXd L = L_mat.transpose();

        int *a;
        hipMalloc((void**)&a, sizeof(int) * data.data.N_x);
        // VectorXi a(data.data.N_x);
        hipStream_t streamPF;
        hipStreamCreateWithFlags(&streamPF, hipStreamNonBlocking);
        __syncthreads();

        // Propagate for each particle (__device__ kernel)
        // This is one-step ahead prediction
        double *x_predicted, *w_x_predicted;
        hipMalloc((void**)&x_predicted, sizeof(double) * data.data.N_x);
        hipMalloc((void**)&w_x_predicted, sizeof(double) * data.data.N_x);

        PropagateState<<<1, data.data.N_x, 0, streamPF>>>(global_state, T_next, x_predicted, w_x_predicted, L.data(), data);
        // Metropolis resampling (__device__ kernel)

        MetropolisResampling<<<1, data.data.N_x, 0, streamPF>>>(global_state, w_x_predicted, data.data.N_x, data.data.B, a);

        // PermutateStatesAndWeights<<<1, data.data.N_x, 0, streamPF>>>(data, x_predicted, w_x_predicted, a);
        PermutateStatesAndWeights<<<1, data.data.N_x, sizeof(double) * data.data.N_x, streamPF>>>(data, x_predicted, w_x_predicted, a); // With shared memory

        // FinalizePF<<<1, 1, 0, cudaStreamTailLaunch>>>
        //         (data, T_next, x_predicted, w_x_predicted, &mlh_hat[tid], &x_hat[data.data.N * tid], &x_particles[data.data.N_x * tid], &w_x_particles[data.data.N_x * tid]);
        FinalizePF<<<1, 1, 0, streamPF>>>
                (data, T_next, x_predicted, w_x_predicted, &mlh_hat[tid], &x_hat[data.data.N * tid], &x_particles[data.data.N_x * tid], &w_x_particles[data.data.N_x * tid]);

        cudaFreePF<<<1, 1, 0, cudaStreamTailLaunch>>>(streamPF, L.data(), a);
    }

}

/**
 * Initialize needed quantities
*/
__global__
void SMC2_init( hiprandState *global_state, 
                const cuData &data, 
                double *theta, 
                double *w_theta, 
                double *f, 
                double *mlh, 
                double *f_particles,    // N_x f-particles for each theta-particle
                double *w_f)            // Weights of the N_x f-particles
{
    // Initialize. For each N_theta theta, do the following:
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // hiprand_init(1234, 0, 0, &state[tid]);
    hiprandState local_state = global_state[tid];
    // Map<MatrixXd> system_x(data.data.X, data.data.N, 1);
    Map<MatrixXd> f_mat(f, data.data.N, data.data.N_theta);
    Map<VectorXd> mlh_vec(mlh, data.data.N_theta);
    Map<MatrixXd> theta_mat(theta, 2, data.data.N_theta);
    Map<MatrixXd> w_theta_mat(w_theta, data.data.N_theta, data.data.N);
    Map<MatrixXd> f_particles_mat(f_particles, data.data.N_x, data.data.N_theta);
    Map<MatrixXd> w_f_mat(w_f, data.data.N_x, data.data.N_theta);
    mlh_vec.setOnes();
    if (tid < data.data.N_theta)
    {
        // Sample N_theta theta-particles and their weight
        // double *theta = new double[2];
        for (int ii = 0; ii < 2; ii++)
        {
            theta_mat(ii, tid) = 1.0 + hiprand_normal_double(&local_state);
        }
        // w_theta_mat.col(0) = VectorXd::Ones(data.data.N_theta) / data.data.N_theta;
        w_theta_mat.setConstant(1.0 / data.data.N_theta);

        // Sample N_theta x0 for each theta-particle
        double *K;
        hipMalloc((void**)&K, sizeof(double) * data.data.N * data.data.N);
        computeKernel(data.data.X, data.data.N, data.data.X, data.data.N, theta[0], theta[1], K);
        // MatrixXd L = K.llt().matrixL();
        MatrixXd L = MatrixXd::Zero(data.data.N, data.data.N);
        cuCholesky(K, data.data.N, L.data());
        VectorXd rand_var(data.data.N);
        for (int kk = 0; kk < data.data.N; kk++)
        {
            rand_var(kk) = hiprand_normal_double(&local_state);
        }
        f_mat(0, tid) = L(0, 0) * rand_var(0);
        __syncthreads();
        // Create the particle ensemble at time=0
        for (int ii = 0; ii < data.data.N_x; ii++)
        {
            f_particles_mat(ii, tid) = f_mat(0, tid) + hiprand_normal_double(&local_state);
        }
        __syncthreads();
        w_f_mat.setConstant(1.0 / data.data.N_x);

        global_state[tid] = local_state;
        hipFree(K);
    }
}

__global__
void PermutateThetaAndWeights(  const cuData &data, 
                                double *theta,          // [2 x N_theta] matrix of parameters a t=T_current
                                double *x_hat,          // [N x N_theta] State trajectory t=1:T_current
                                double *mlh,            // [N_theta x 1] Marginal LH at current time
                                double *x_particles,    // [N_x x N_theta] N_x particles for each theta
                                double *w_x_particles,  // [N_x x N_theta] Weights
                                const int *a            // [N_theta x 1] Ancestors
                                )
{
    extern __shared__ double tmp_buffer[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("blockDim = %d; blockIdx = %d; threadIdx = %d; tid = %d\n", blockDim.x, blockIdx.x, threadIdx.x, tid);
    if (tid < data.data.N_theta)
    {
        // For each theta, copy the values of input args and then shuffle them using vector a.
        #if VERBOSE
        if (tid == 0)
        {
            printf("[PERM_THETA] Permutate tid = %d\n", tid);
        }
        #endif
        // printf("a_theta contains: %d %d %d %d\n", a[0], a[1], a[2], a[3]);
        
        Map<VectorXd> mlh_vec(mlh, data.data.N_theta);
        if (mlh_vec.sum() < 1e-100)
        {
            mlh[tid] = 1.0 / data.data.N_theta;
            // printf("Re-initializing theta-weights...\n");
        }
        else
        {
            // hipMalloc((void**)&mlh_old, sizeof(double) * data.data.N_theta);
            memcpy(&tmp_buffer, mlh, sizeof(double) * data.data.N_theta);
            memcpy(&mlh[tid], &tmp_buffer[a[tid]], sizeof(double));
        }
        // printf("[PERM_THETA] a contains: %d %d %d %d.\tmlh[tid]: %e; sum(mlh): %e\n", a[0], a[1], a[2], a[3], mlh[tid], mlh_vec.sum());

        memcpy(&tmp_buffer, theta, sizeof(double) * 2 * data.data.N_theta);
        memcpy(&theta[2*tid], &tmp_buffer[2*a[tid]], sizeof(double) * 2);

        memcpy(&tmp_buffer, x_hat, sizeof(double) * data.data.N * data.data.N_theta);
        memcpy(&x_hat[data.data.N * tid], &tmp_buffer[data.data.N * a[tid]], sizeof(double) * data.data.N);
        
        memcpy(&tmp_buffer, x_particles, sizeof(double) * data.data.N_x * data.data.N_theta);
        memcpy(&x_particles[data.data.N_x*tid], &tmp_buffer[data.data.N_x*a[tid]], sizeof(double) * data.data.N_x);

        memcpy(&tmp_buffer, w_x_particles, sizeof(double) * data.data.N_x * data.data.N_theta);
        memcpy(&w_x_particles[data.data.N_x*tid], &tmp_buffer[data.data.N_x*a[tid]], sizeof(double) * data.data.N_x);

    }
}

__global__
void NormalizeWeights(  const cuData &data,
                        const double *mlh_hat,      // [N_theta x 1] Marginal LH at current time
                        double *w_theta)            // [N_theta x 1] (Normalized) theta weights at time T_next
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < data.data.N_theta)
    {
        #if VERBOSE
        if (tid == 0)
        {
            printf("[NW] tid: %d\n", tid);
        }
        #endif
        Map<const::VectorXd> mlh_hat_vec(mlh_hat, data.data.N_theta);
        double sum_mlh = mlh_hat_vec.sum();
        w_theta[tid] = mlh_hat_vec(tid) / sum_mlh;
    }
}

__global__
void SMC2run(   hiprandState_t *global_state_theta,
                hiprandState_t *global_state_x,
                const cuData &data,
                Graph &local_path_graph,
                double *theta,                      // [2 x N_theta]
                double *w_theta,                    // [N_theta x N]
                double *mlh,                        // [N_theta x 1]
                double *f_hat,                      // [N x N_theta]
                double *f_particles,                // [N_x x N_theta]
                double *w_f                         // [N_x x N_theta]
                )
{
    // Update current time and check if it is bigger than the boundaries
    local_path_graph.current = local_path_graph.current + local_path_graph.direction;
    if (local_path_graph.direction == 1)
    {
        if (local_path_graph.current > local_path_graph.last)
        {
            // Reached boundaries of graph
            return;
        }
    }
    if (local_path_graph.direction == -1)
    {
        if (local_path_graph.current < local_path_graph.last)
        {
            return;
        }
    }

    // Otherwise, recursively call this function until boundaries are reached
    printf("\x1B[34m============================== Time: %d ==============================\n\x1B[0m", local_path_graph.current);
    hipStream_t streamSMCrun;
    hipStreamCreateWithFlags(&streamSMCrun, hipStreamNonBlocking);
    ParticleFilter<<<data.data.N_theta, 1, 0, streamSMCrun>>>
            (theta, local_path_graph.current, data, global_state_x, mlh, f_hat, f_particles, w_f);
    NormalizeWeights<<<data.data.N_theta, 1, 0, streamSMCrun>>>
            (data, mlh, &w_theta[local_path_graph.current * data.data.N_theta]);

    int *ancestors;
    hipMalloc((void**)&ancestors, sizeof(int) * data.data.N_theta);
    MetropolisResampling<<<1, data.data.N_theta, 0, streamSMCrun>>>
            (global_state_theta, &w_theta[local_path_graph.current * data.data.N_theta], data.data.N_theta, data.data.B, ancestors);
    // MetropolisResampling<<<data.N_theta, 1>>>(devStates_theta, &dev_w_theta[t * dev_data->data.N_theta], dev_data->data.N_theta, B, dev_ancestors);
    // VectorXi h_a(data.N_theta);
    // CUDA_CHECK(hipMemcpy(h_a.data(), dev_ancestors, sizeof(int) * data.N_theta, hipMemcpyDeviceToHost));
    // std::cout << "Ancestors:\n" << h_a.transpose() << std::endl;

    size_t dim_buffer = max(data.data.N * data.data.N_theta, data.data.N_theta * data.data.N_x);
    PermutateThetaAndWeights<<<1, data.data.N_theta, dim_buffer * sizeof(double), streamSMCrun>>>
                (data, theta, f_hat, mlh, f_particles, w_f, ancestors); // With shared memory


    // CUDA_CHECK(hipMemcpy(h_theta.data(), dev_theta, sizeof(double) * 2 * data.N_theta, hipMemcpyDeviceToHost));
    // std::cout << "Theta permutated:\n" << h_theta << std::endl;

    // MarginalMetropolisHastings<<<1, data.N_theta>>>(devStates_theta, devStates_x, t, dev_theta, dev_f, dev_mlh, dev_f_particles, dev_w_f, *dev_data);
    MarginalMetropolisHastings<<<data.data.N_theta, 1, 0, streamSMCrun>>>
            (global_state_theta, global_state_x, local_path_graph, theta, f_hat, mlh, f_particles, w_f, data);

    SMC2run<<<1, 1, 0, streamSMCrun>>>
            (global_state_theta, global_state_x, data, local_path_graph, theta, w_theta, mlh, f_hat, f_particles, w_f);

    cudaFreePF<<<1, 1, 0, cudaStreamTailLaunch>>>(streamSMCrun, nullptr, ancestors);
}

/**
 * Sequential Monte Carlo² (SMC²) algorithm
*/
void SMC2(const Data &data)
{
    hiprandState *devStates_theta, *devStates_x;
    // int totalThreads = 256;// data.N_theta * data.N_x;// 256;
    CUDA_CHECK(hipMalloc((void **)&devStates_theta, data.N_theta * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc((void **)&devStates_x, data.N_x * sizeof(hiprandState)));
    double *dev_theta, *dev_w_theta, *dev_f, *dev_mlh, *dev_f_particles, *dev_w_f;
    int *dev_ancestors;
    CUDA_CHECK(hipMalloc((void**)&dev_theta, sizeof(double) * 2 * data.N_theta));
    CUDA_CHECK(hipMalloc((void**)&dev_w_theta, sizeof(double) * data.N_theta * data.N));
    CUDA_CHECK(hipMalloc((void**)&dev_f, sizeof(double) * data.N_theta * data.N));
    CUDA_CHECK(hipMalloc((void**)&dev_mlh, sizeof(double) * data.N_theta));
    CUDA_CHECK(hipMalloc((void**)&dev_f_particles, sizeof(double) * data.N_x * data.N_theta));
    CUDA_CHECK(hipMalloc((void**)&dev_w_f, sizeof(double) * data.N_x * data.N_theta));
    CUDA_CHECK(hipMalloc((void**)&dev_ancestors, sizeof(int) * data.N_theta));
    cuData *dev_data = new cuData;
    dev_data->data = data;
    CUDA_CHECK(hipMallocManaged((void**)&(dev_data->data.X), sizeof(double) * data.N));
    std::copy(data.X, data.X + sizeof(double) * data.N, dev_data->data.X);
    CUDA_CHECK(hipMallocManaged((void**)&(dev_data->data.Y), sizeof(double) * data.N));
    std::copy(data.Y, data.Y + sizeof(double) * data.N, dev_data->data.Y);
    Graph *dev_initial_graph = new Graph;

    int B = data.B;
    MatrixXd w_theta = MatrixXd::Zero(data.N_theta, data.N);
    MatrixXd theta = MatrixXd::Zero(2, data.N_theta);
    MatrixXd f = MatrixXd::Zero(data.N, data.N_theta);
    Map<VectorXd> system_x(data.X, data.N);
    Map<VectorXd> system_y(data.Y, data.N);

    // Initialize
    setup_curand_theta<<<1, data.N_theta>>>(devStates_theta);
    setup_curand_x<<<1, data.N_x>>>(devStates_x);
    SMC2_init<<<1, data.N_theta>>>(devStates_theta, *dev_data, dev_theta, dev_w_theta, dev_f, dev_mlh, dev_f_particles, dev_w_f);
    hipDeviceSynchronize();
    // MatrixXd h_f_parts(data.N_x, data.N_theta);
    // CUDA_CHECK(hipMemcpy(h_f_parts.data(), dev_f_particles, sizeof(double) * data.N_x * data.N_theta, hipMemcpyDeviceToHost));
    // std::cout << "dev_w_f after initialization:\n" << h_f_parts << std::endl;
    VectorXd x_final(data.N);
    #if PLOT
    plt::figure(1);
    plt::figure(2);
    #endif
    std::cout << "Init done!\n";
    // Cycle through time
    // for (int t = 0; t < data.N - 1; t++)
    // {

        int t = trunc(data.N / 2);
        dev_initial_graph->first = t;
        dev_initial_graph->current = t;

        MatrixXd h_theta(2, data.N_theta), h_f(data.N, data.N_theta);
        // CUDA_CHECK(hipMemcpy(h_theta.data(), dev_theta, sizeof(double) * 2 * data.N_theta, hipMemcpyDeviceToHost));
        // std::cout << "Theta non-permutated:\n" << h_theta << std::endl;
        std::cout << "\x1B[34m============================== Time: " << t << " ==============================\n\x1B[0m";

        MetropolisResampling<<<1, data.N_theta>>>(devStates_theta, &dev_w_theta[t * dev_data->data.N_theta], dev_data->data.N_theta, B, dev_ancestors);
        // MetropolisResampling<<<data.N_theta, 1>>>(devStates_theta, &dev_w_theta[t * dev_data->data.N_theta], dev_data->data.N_theta, B, dev_ancestors);
        // VectorXi h_a(data.N_theta);
        // CUDA_CHECK(hipMemcpy(h_a.data(), dev_ancestors, sizeof(int) * data.N_theta, hipMemcpyDeviceToHost));
        // std::cout << "Ancestors:\n" << h_a.transpose() << std::endl;

        // PermutateThetaAndWeights<<<1, data.N_theta>>>(*dev_data, dev_theta, dev_f, dev_mlh, dev_f_particles, dev_w_f, dev_ancestors);
        // PermutateThetaAndWeights<<<data.N_theta, 1>>>(*dev_data, dev_theta, dev_f, dev_mlh, dev_f_particles, dev_w_f, dev_ancestors);
        size_t dim_buffer = max(data.N * data.N_theta, data.N_theta * data.N_x);
        PermutateThetaAndWeights<<<1, data.N_theta, dim_buffer * sizeof(double)>>>
                    (*dev_data, dev_theta, dev_f, dev_mlh, dev_f_particles, dev_w_f, dev_ancestors); // With shared memory


        // CUDA_CHECK(hipDeviceSynchronize());
        // CUDA_CHECK(hipMemcpy(h_theta.data(), dev_theta, sizeof(double) * 2 * data.N_theta, hipMemcpyDeviceToHost));
        // std::cout << "Theta permutated:\n" << h_theta << std::endl;

        // MarginalMetropolisHastings<<<1, data.N_theta>>>(devStates_theta, devStates_x, t, dev_theta, dev_f, dev_mlh, dev_f_particles, dev_w_f, *dev_data);
        MarginalMetropolisHastings<<<data.N_theta, 1>>>(devStates_theta, devStates_x, *dev_initial_graph, dev_theta, dev_f, dev_mlh, dev_f_particles, dev_w_f, *dev_data);
        // CUDA_CHECK(hipDeviceSynchronize());
        size_t initial_nodes = 2;   // For now, only 2 directions (from center to left + from center to right)
        Graph *dev_graph = new Graph[initial_nodes];
        CUDA_CHECK(hipMallocManaged((void**)&dev_graph, initial_nodes * sizeof(Graph)));
        dev_graph[0].first      = t;
        dev_graph[1].first      = t;
        dev_graph[0].last       = 0;
        dev_graph[1].last       = data.N - 1;
        dev_graph[0].direction  = -1;
        dev_graph[1].direction  = 1;
        dev_graph[0].current    = t;
        dev_graph[1].current    = t;
        hipStream_t streamSMC2[initial_nodes];

        for (int n = 0; n < initial_nodes; n++)
        {
            // Start kernels
            CUDA_CHECK(hipStreamCreateWithFlags(&streamSMC2[n], hipStreamNonBlocking));
            SMC2run<<<1, 1, 0, streamSMC2[n]>>>
                    (devStates_theta, devStates_x, *dev_data, dev_graph[n], dev_theta, dev_w_theta, dev_mlh, dev_f, dev_f_particles, dev_w_f);

        }
        CUDA_CHECK(hipDeviceSynchronize());
        /*
        CUDA_CHECK(hipDeviceSynchronize());

        VectorXd h_mlh(data.N_theta);
        CUDA_CHECK(hipMemcpy(h_mlh.data(), dev_mlh, sizeof(double) * data.N_theta, hipMemcpyDeviceToHost));
        std::cout << "average mlh at time " << T_next << " is:\n" << h_mlh.mean() << std::endl;
        // CUDA_CHECK(hipMemcpy(h_f.data(), dev_f, sizeof(double) * data.N * data.N_theta, hipMemcpyDeviceToHost));
        // std::cout << "f full:\n" << h_f << std::endl;
        CUDA_CHECK(hipMemcpy(f.data(), dev_f, sizeof(double) * data.N * data.N_theta, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(theta.data(), dev_theta, sizeof(double) * 2 * data.N_theta, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(w_theta.data(), dev_w_theta, sizeof(double) * data.N * data.N_theta, hipMemcpyDeviceToHost));

        VectorXd x_hat_all = (w_theta.leftCols(T_next+1).transpose().array() * f.topRows(T_next+1).array()).rowwise().sum();
        std::cout << "x_hat:\n" << x_hat_all.transpose() << std::endl;
        VectorXd theta_hat_all(2);
        theta_hat_all(0) = (w_theta.col(T_next).array() * theta.row(0).transpose().array()).sum();
        theta_hat_all(1) = (w_theta.col(T_next).array() * theta.row(1).transpose().array()).sum();
        std::cout << "Theta is:\n" << theta_hat_all.transpose() << std::endl;*/

        #if PLOT
        // plt::figure(1);
        // plt::clf();
        // plt::plot(system_x.head(T_next), system_y.head(T_next), "k+");
        // plt::plot(system_x.head(T_next), x_hat_all);
        // plt::show(false);

        plt::figure(1);
        // plt::clf();
        plt::plot(system_x.segment(t, 2), system_y.segment(t, 2), "k+");
        plt::plot(system_x.segment(t, 2), x_hat_all.segment(t, 2), "b");
        plt::show(false);

        plt::figure(2);
        VectorXd t_vec(1), theta_1(1), theta_2(1);
        t_vec << t; theta_1 << pow(theta_hat_all(0), 2); theta_2 << pow(theta_hat_all(1), 2);

        plt::plot(t_vec, theta_1, "bo");
        plt::plot(t_vec, theta_2, "ro");
        plt::show(false);
        plt::pause(0.05);

        x_final.head(T_next) = x_hat_all;
        #endif
        

        
    // }
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    #if FINAL_PLOT
    CUDA_CHECK(hipMemcpy(f.data(), dev_f, sizeof(double) * data.N * data.N_theta, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(w_theta.data(), dev_w_theta, sizeof(double) * data.N * data.N_theta, hipMemcpyDeviceToHost));

    VectorXd x_hat_all = (w_theta.transpose().array() * f.array()).rowwise().sum();
    std::cout << "x_hat:\n" << x_hat_all.transpose() << std::endl;
    
    plt::figure(3);
    plt::plot(system_x, system_y, "k+");
    // plt::plot(system_x, x_final);
    plt::plot(system_x, x_hat_all);
    plt::show(true);
    #endif

}