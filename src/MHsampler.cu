#include "hip/hip_runtime.h"
#include "MHsampler.hpp"

// using namespace Eigen;

/*
VectorXd mvn_sampler(hiprandGenerator_t &gen, int num_samples, VectorXd &mean, MatrixXd &cov)
{
    // Multivariate normal sampler using cuRAND

    float *samples;
    CUDA_CHECK(hipMalloc((void**)&samples, sizeof(float) * num_samples));
    hiprandGenerateNormal(gen, samples, num_samples, 0.0, 1.0);

    VectorXf rand_data(num_samples);
    CUDA_CHECK(hipMemcpy(rand_data.data(), samples, sizeof(float) * num_samples, hipMemcpyDeviceToHost));

    // Sample from the multivariate normal
    LLT<MatrixXd> llt_dec(cov);
    MatrixXd L = cov.llt().matrixL();
    VectorXd out = mean + L * rand_data.cast<double>();
    return out;
};

VectorXd mvn_sampler_double(hiprandGenerator_t &gen, int num_samples, VectorXd &mean, MatrixXd &cov)
{
    // Multivariate normal sampler using cuRAND

    double *samples;
    CUDA_CHECK(hipMalloc((void**)&samples, sizeof(double) * num_samples));
    hiprandGenerateNormalDouble(gen, samples, num_samples, 0.0, 1.0);

    VectorXd rand_data(num_samples);
    CUDA_CHECK(hipMemcpy(rand_data.data(), samples, sizeof(double) * num_samples, hipMemcpyDeviceToHost));

    // Sample from the multivariate normal
    LLT<MatrixXd> llt_dec(cov);
    MatrixXd L = cov.llt().matrixL();
    VectorXd out = mean + L * rand_data;
    return out;
};

VectorXd uniform_sampler(hiprandGenerator_t &gen, int num_samples)
{
    float *samples;
    CUDA_CHECK(hipMalloc((void**)&samples, sizeof(float) * num_samples));
    hiprandGenerateUniform(gen, samples, num_samples);

    VectorXf rand_data(num_samples);
    CUDA_CHECK(hipMemcpy(rand_data.data(), samples, sizeof(float) * num_samples, hipMemcpyDeviceToHost));
    // CUDA_CHECK(hipMemcpyAsync(rand_data.data(), samples, sizeof(float) * num_samples, hipMemcpyDeviceToHost, stream));
    return rand_data.cast<double>();
}

VectorXd uniform_sampler_double(hiprandGenerator_t &gen, int num_samples)
{
    double *samples;
    CUDA_CHECK(hipMalloc((void**)&samples, sizeof(double) * num_samples));
    hiprandGenerateUniformDouble(gen, samples, num_samples);

    VectorXd rand_data(num_samples);
    CUDA_CHECK(hipMemcpy(rand_data.data(), samples, sizeof(double) * num_samples, hipMemcpyDeviceToHost));
    // CUDA_CHECK(hipMemcpyAsync(rand_data.data(), samples, sizeof(float) * num_samples, hipMemcpyDeviceToHost, stream));
    return rand_data;
}


VectorXd uni_to_multivariate(const VectorXf &random_samples, const VectorXd &mean, const MatrixXd &cov)
{
    LLT<MatrixXd> llt_dec(cov);
    MatrixXd L = cov.llt().matrixL();
    VectorXd out = mean + L * random_samples.cast<double>();
    return out;
}

VectorXd uni_to_multivariate_double(const VectorXd &random_samples, const VectorXd &mean, const MatrixXd &cov)
{
    LLT<MatrixXd> llt_dec(cov);
    MatrixXd L = cov.llt().matrixL();
    VectorXd out = mean + L * random_samples;
    return out;
}*/

MatrixXd MHsampler(Data &data, Distribution &prior, Distribution &likelihood, Distribution &proposal, MHoptions opts)
{
    //
    int N = data.x_train.rows(); // # of training points
    hiprandGenerator_t gen;

    // hipStream_t stream_1;
    // hipStreamCreate(&stream_1);

    // float *samples;
    // // hipMalloc((void**)samples, sizeof(float) * data.x_train.size() * opts.max_iterations);
    // CUDA_CHECK(hipMalloc((void**)&samples, sizeof(float) * N * (opts.max_iterations + opts.burnin)));
    // hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
    // // hiprandGenerateNormal(gen, samples, N, 0.0, 1.0);
    // hiprandStatus_t status = hiprandGenerateNormal(gen, samples, N * (opts.max_iterations + opts.burnin), 0.0, 1.0);
    // std::cout << "hiprand status: " << status << std::endl;

    // VectorXf rand_data(N * (opts.max_iterations + opts.burnin));
    // CUDA_CHECK(hipMemcpy(rand_data.data(), samples, sizeof(float) * N * (opts.max_iterations + opts.burnin), hipMemcpyDeviceToHost));
    // CUDA_CHECK(hipMemcpyAsync(rand_data.data(), samples, sizeof(float) * N * (opts.max_iterations + opts.burnin), hipMemcpyDeviceToHost, stream_1));

    // double *samples;
    float *samples;
    // hipMalloc((void**)samples, sizeof(float) * data.x_train.size() * opts.max_iterations);
    // CUDA_CHECK(hipMalloc((void**)&samples, sizeof(double) * N * (opts.max_iterations + opts.burnin)));
    CUDA_CHECK(hipMalloc((void**)&samples, sizeof(float) * N * (opts.max_iterations + opts.burnin)));
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, rand() * 25ULL /*1234ULL*/);
    hiprandStatus_t status = hiprandGenerateNormal(gen, samples, N * (opts.max_iterations + opts.burnin), 0.0, 1.0);
    // hiprandStatus_t status = hiprandGenerateNormalDouble(gen, samples, N * (opts.max_iterations + opts.burnin), 0.0, 1.0);
    std::cout << "hiprand status: " << status << std::endl;

    // VectorXd rand_data(N * (opts.max_iterations + opts.burnin));
    VectorXf rand_data(N * (opts.max_iterations + opts.burnin));
    // CUDA_CHECK(hipMemcpy(rand_data.data(), samples, sizeof(double) * N * (opts.max_iterations + opts.burnin), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(rand_data.data(), samples, sizeof(float) * N * (opts.max_iterations + opts.burnin), hipMemcpyDeviceToHost));

    // Sample from the multivariate prior
    VectorXd f = mvn_sampler(gen, N, prior.mean, prior.covariance);
    // VectorXd f = mvn_sampler_double(gen, N, prior.mean, prior.covariance);
    // VectorXd f = prior.mean + prior.covariance.llt().matrixL() * rand_data.cast<double>();

    // Sample all the acceptance thresholds
    VectorXd acceptanceThr = log(uniform_sampler(gen, opts.max_iterations + opts.burnin).array());
    // VectorXd acceptanceThr = log(uniform_sampler_double(gen, opts.max_iterations + opts.burnin).array());

    int accepted_samples = 0;
    int idx = 0;
    MatrixXd MHsamples((int)floor(opts.max_iterations / opts.store_after), N);
    double sigma_n = likelihood.covariance(0,0);

    std::cout << "Iteration:\n";
    for (int i = 0; i < opts.max_iterations + opts.burnin; i++)
    {
        // VectorXd fnew = mvn_sampler(gen, N, proposal.mean, proposal.covariance);
        VectorXd fnew = uni_to_multivariate(rand_data.segment(i*N, N), proposal.mean, proposal.covariance);
        // VectorXd fnew = uni_to_multivariate_double(rand_data.segment(i*N, N), proposal.mean, proposal.covariance);
        
        // Evaluate likelihoods
        double lh_new = -0.5 * log( pow(2 * M_PI, N) * sigma_n) - 
                    0.5 / sigma_n * pow((data.y_train - fnew).array(), 2).sum(); // (data.y_train - fnew) * (data.y_train - fnew).transpose();

        double lh_old = -0.5 * log( pow(2 * M_PI, N) * sigma_n) - 
                    0.5 / sigma_n * pow((data.y_train - f).array(), 2).sum();

        double acceptanceProb = lh_new - lh_old;
        if (min(acceptanceProb, 0.0 ) > acceptanceThr(i))
        {
            f = fnew;
            accepted_samples++;
        }

        if (i > opts.burnin && i % opts.store_after == 0)
        {
            MHsamples.row(idx++) = f;
        }
        // printf("\e[1K\e[1G%d", i);
        // std::cout << "Iteration " << i;
        
    }
    std::cout << "\nAcceptance rate: " << accepted_samples / (float)opts.max_iterations * 100 << "%." << std::endl;
    std::cout << "Sample size " << MHsamples.rows() << "x" << MHsamples.cols() << std::endl;
    return MHsamples;

}